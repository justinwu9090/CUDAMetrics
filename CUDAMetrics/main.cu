#include "hip/hip_runtime.h"
﻿// cuda
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

// more standard libraries
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <map>
#include <string>

using namespace std;

// local
#include "helper.cuh"
#include "add.cuh"

int main()
{
	hipDeviceProp_t props = dumpDeviceProperties(true);

	// theoretical max occupancy threads based on specific CUDA device properties. # SM's X Max Blocks/SM X maxThreads per SM
	int maxoccupancythreads = props.multiProcessorCount * props.maxBlocksPerMultiProcessor * props.maxThreadsPerMultiProcessor;
	cout << "maxoccupancythreads: " << maxoccupancythreads << endl;


	int arraySize = maxoccupancythreads;

	ofstream myfile;
	myfile.open("test.csv", std::ios_base::app);

	// Add vectors in parallel - call AddKernel Wrapper
	//vector<unsigned int> sizes = { arraySize * 4, arraySize * 64, arraySize * 128, arraySize * 256, arraySize * 512, arraySize * 1024, arraySize * 2048 };
	vector<int> sizes({ arraySize * 4, arraySize * 64, arraySize * 128, arraySize * 256, arraySize * 512, arraySize * 1024, arraySize * 2048 });
	//vector<int> sizes({ arraySize * 4, arraySize * 64});
	for (auto arrsize : sizes)
	{

		auto results = addKernelWrapper(arrsize, false);

		// add to results csv.
		myfile << results["name"] << "," << results["grid_dim"] << "," << results["block_dim"] << "," << results["array_size"] << "," << results["duration_us"] << "," << endl;

		results = addSequentialWrapper(arrsize, false);
		myfile << results["name"] << "," << results["grid_dim"] << "," << results["block_dim"] << "," << results["array_size"] << "," << results["duration_us"] << "," << endl;
	}
	myfile.close();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	errCatch(hipDeviceReset());

	return 0;
}


