#include "hip/hip_runtime.h"
﻿// cuda
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>

// more standard libraries
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
using namespace std;

// timekeeping
#include <chrono>
using std::chrono::high_resolution_clock;
using std::chrono::duration_cast;
using std::chrono::duration;

__host__ void errCatch(hipError_t);
__host__ void addWithCuda(vector<int>& c, vector<int>& a, vector<int>& b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

hipDeviceProp_t dumpDeviceProperties()
{
	int deviceID;
	hipDeviceProp_t props;

	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&props, deviceID);
	return props;
}

void simpleAdd(int arraySize)
{
	ofstream myfile;
	myfile.open("test.csv");
	vector<int> a(arraySize, 1);
	vector<int> b(arraySize, 1);
	vector<int> c(arraySize, 0);
	addWithCuda(c, a, b, arraySize);
}
int main()
{
	hipDeviceProp_t props = dumpDeviceProperties();
	cout << "GPU: " << props.name << endl;
	cout << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor << endl;
	cout << "maxBlocksPerMultiProcessor: " << props.maxBlocksPerMultiProcessor << endl;
	cout << "multiProcessorCount: " << props.multiProcessorCount << endl;

	int maxoccupancythreads = props.multiProcessorCount * props.maxBlocksPerMultiProcessor * props.maxThreadsPerMultiProcessor;
	cout << "maxoccupancythreads: " << maxoccupancythreads << endl;


	const unsigned int arraySize = maxoccupancythreads;
	

	// Add vectors in parallel.
	simpleAdd(arraySize);
	simpleAdd(arraySize*2);
	simpleAdd(arraySize*3);
	
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	errCatch(hipDeviceReset());

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
__host__ void addWithCuda(vector<int>& c, vector<int>& a, vector<int>& b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.

	errCatch(hipSetDevice(0));

	// Allocate GPU buffers for three vectors (two input, one output)    .
	errCatch(hipMalloc((void**)&dev_c, size * sizeof(int)));
	errCatch(hipMalloc((void**)&dev_a, size * sizeof(int)));
	errCatch(hipMalloc((void**)&dev_b, size * sizeof(int)));

	const int ROUNDS = 5;
	for (int i = 0; i < ROUNDS; i++)
	{

		// Copy input vectors from host memory to GPU buffers.
		errCatch(hipMemcpy(dev_a, a.data(), size * sizeof(int), hipMemcpyHostToDevice));
		errCatch(hipMemcpy(dev_b, b.data(), size * sizeof(int), hipMemcpyHostToDevice));

		auto t1 = high_resolution_clock::now();
		// Launch a kernel on the GPU with one thread for each element.
		dim3 gridDimInBlocks(ceil((float)size / 32), 1, 1);
		dim3 blockDimInThreads(32, 1, 1);
		addKernel << <gridDimInBlocks, blockDimInThreads >> > (dev_c, dev_a, dev_b);
		auto t2 = high_resolution_clock::now();

		/* Getting number of milliseconds as an integer. */
		auto us_int = duration_cast<std::chrono::microseconds>(t2 - t1);

		// Check for any errors launching the kernel
		errCatch(hipGetLastError());

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		errCatch(hipDeviceSynchronize());

		// Copy output vector from GPU buffer to host memory.
		errCatch(hipMemcpy(&c[0], dev_c, size * sizeof(int), hipMemcpyDeviceToHost));

		for (int i = 0; i < c.size(); i++)
		{
			int ans = c[i];
			if (ans != a[0] + b[0])
			{
				printf("output wrong here\n");
			}
		}

		printf("%d us\n", (int) us_int.count());
	}

	//Error:
	errCatch(hipFree(dev_c));
	errCatch(hipFree(dev_a));
	errCatch(hipFree(dev_b));
}

// Catches errors returned from CUDA functions
__host__ void errCatch(hipError_t err) {
	if (err != hipSuccess) {
		cout << hipGetErrorString(err) << endl;
		exit(EXIT_FAILURE);
	}
}

// Returns the size in bytes of any type of vector
template<typename T>
size_t vBytes(const typename vector<T>& v) {
	return sizeof(T) * v.size();
}
