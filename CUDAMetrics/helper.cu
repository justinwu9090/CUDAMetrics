// cuda
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>

// more standard libraries
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <map>
#include <string>

using namespace std;

// Catches errors returned from CUDA functions
__host__ void errCatch(hipError_t err) {
	if (err != hipSuccess) {
		cout << hipGetErrorString(err) << endl;
		exit(EXIT_FAILURE);
	}
}

// Returns the size in bytes of any type of vector
template<typename T>
size_t vBytes(const typename vector<T>& v) {
	return sizeof(T) * v.size();
}

hipDeviceProp_t dumpDeviceProperties(bool printout = true)
{
	int deviceID;
	hipDeviceProp_t props;

	hipGetDevice(&deviceID);
	hipGetDeviceProperties(&props, deviceID);
	if (printout)
	{
		cout << "GPU: " << props.name << endl;
		cout << "maxThreadsPerMultiProcessor: " << props.maxThreadsPerMultiProcessor << endl;
		cout << "maxBlocksPerMultiProcessor: " << props.maxBlocksPerMultiProcessor << endl;
		cout << "multiProcessorCount: " << props.multiProcessorCount << endl;
	}

	return props;
}